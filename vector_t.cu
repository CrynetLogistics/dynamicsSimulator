#include "hip/hip_runtime.h"
#include "vector_t.h"

__host__ __device__ vector_t::vector_t(void)
{
}

__host__ __device__ vector_t::vector_t(vertex_t origin, vertex_t destination){
	xt = destination.x - origin.x;
	yt = destination.y - origin.y;
	zt = destination.z - origin.z;
}

__host__ __device__ vector_t::vector_t(float xt, float yt, float zt)
{
	this->xt = xt;
	this->yt = yt;
	this->zt = zt;
}

__host__ __device__ float vector_t::calculateDistance(float t){
	float xdist = xt*t;
	float ydist = yt*t;
	float zdist = zt*t;
	return sqrt(xdist*xdist+ydist*ydist+zdist*zdist);
}

__host__ __device__ float vector_t::directionDotProduct(vector_t dotterand){
	return xt*dotterand.xt + yt*dotterand.yt + zt*dotterand.zt;
}

__host__ __device__ vector_t vector_t::directionCrossProduct(vector_t crosserand){
	vector_t vector;
	vector.xt = yt*crosserand.zt - zt*crosserand.yt;
	vector.yt = zt*crosserand.xt - xt*crosserand.zt;
	vector.zt = xt*crosserand.yt - yt*crosserand.xt;
	return vector;
}

__host__ __device__ float vector_t::directionMagnitude(void){
	return sqrt(xt*xt+yt*yt+zt*zt);
}

__host__ __device__ vector_t::~vector_t(void)
{
}
